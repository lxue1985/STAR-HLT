#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>


#include <cuda_gl_interop.h>
#include "lib_kernel.cuh"


__global__ void test(float4* position, int2*hash,int*map,int* manager, float *out, int *number){


#define DIM 24
#define MAXTRACK 500

 int index;

 index = blockIdx.x*blockDim.x + threadIdx.x;
 

__shared__ int sindex[DIM];
__shared__ int sblockIdx[DIM];

sindex[threadIdx.x]=index;
sblockIdx[threadIdx.x]=manager[blockIdx.x+30];

if(sindex[threadIdx.x]>=number[sblockIdx[threadIdx.x]]) return;

small_FtfTrack* out_track=  (small_FtfTrack*)(out+sblockIdx[threadIdx.x]*sizeof(small_FtfTrack)*MAXTRACK);

// __shared__   int  track_index[DIM];


__shared__  small_FtfTrack localtrack[DIM];
__shared__  small_FtfTrack* track[DIM];


__shared__   int  hashIndex[DIM];


__shared__   int  candidate[DIM];
__shared__   int  cnum[DIM];


	__shared__   int  cei[DIM];
	__shared__   int  cpi[DIM];
	__shared__   int  cri[DIM];

   __shared__   int  loop_eta[9];
   __shared__   int  loop_phi[9];
__shared__   int  rhoi[DIM];
__shared__   int  phii[DIM];
__shared__   int  etai[DIM];
__shared__   int  miss[DIM];
__shared__   int  ccri[DIM];

__shared__   int  result[DIM];	

__shared__   int  curmap[DIM];
__shared__   int  curpos[DIM];	
__shared__   small_FtfHit selected_hit[DIM];
__shared__   int  map_index[DIM];
__shared__   int  pos_index[DIM];
__shared__   int  ccei[DIM];
__shared__   int  ccpi[DIM];
__shared__   int  cchashIndex[DIM];
__shared__   int  hit_num[DIM];
__shared__   int  start[DIM];
__shared__   int  hl[DIM];

__shared__   float  dx[DIM];
__shared__   float  dy[DIM];

 __shared__   float  xyChi2[DIM];
 __shared__   float  szChi2[DIM];
__shared__   float  lxyChi2[DIM];
// __shared__   float  lszChi2[DIM];
// __shared__   float  slocal[DIM];


		

__shared__   float  r2[DIM];

__shared__   float  invR2[DIM];


   
   loop_eta[0]=0;
   loop_eta[1]=0;
   loop_eta[2]=0;
   loop_eta[3]=-1;
   loop_eta[4]=-1;
   loop_eta[5]=-1;
   loop_eta[6]=1;
   loop_eta[7]=1;
   loop_eta[8]=1;
   

	loop_phi[0]=0;   
	loop_phi[1]=-1;   
	loop_phi[2]=1;   
	loop_phi[3]=0;   
	loop_phi[4]=-1;   
	loop_phi[5]=1;   
	loop_phi[6]=0;   
	loop_phi[7]=-1;   
	loop_phi[8]=1;   
   
   
   phii[threadIdx.x]=sindex[threadIdx.x]/41;
   etai[threadIdx.x]=sindex[threadIdx.x]%41;
   
   
   do{
   
   
   cnum[threadIdx.x]=atomicAdd(&map[sblockIdx[threadIdx.x]],1);
   
   
   if(cnum[threadIdx.x]>number[sblockIdx[threadIdx.x]]) break;
   if(position[(sblockIdx[threadIdx.x])*max_hit+cnum[threadIdx.x]].w<0) continue;


   candidate[threadIdx.x]=cnum[threadIdx.x];

   
   cnum[threadIdx.x]=position[(sblockIdx[threadIdx.x])*max_hit+cnum[threadIdx.x]].w;
   
   rhoi[threadIdx.x]=cnum[threadIdx.x]/451;
   
   if(rhoi[threadIdx.x]<7) break;
   
   cnum[threadIdx.x]=cnum[threadIdx.x]%451;

   phii[threadIdx.x]=cnum[threadIdx.x]/41;
   etai[threadIdx.x]=cnum[threadIdx.x]%41;   
 
				// for( rhoi[threadIdx.x]=45;rhoi[threadIdx.x]>0;rhoi[threadIdx.x]--){
				// for( phii[threadIdx.x]=10;phii[threadIdx.x]>0;phii[threadIdx.x]--){	
				// for( etai[threadIdx.x]=40;etai[threadIdx.x]>0;etai[threadIdx.x]--){



 	     hashIndex[threadIdx.x] =  (rhoi[threadIdx.x])  * (41*11) + 
        (phii[threadIdx.x]) * 41 + (etai[threadIdx.x]) ;





 //gmap[threadIdx.x]=map[hashIndex[threadIdx.x]];
 //ghit_num[threadIdx.x]=hash[(sblockIdx[threadIdx.x])*HASHSIZE+hashIndex[threadIdx.x]].y-hash[(sblockIdx[threadIdx.x])*HASHSIZE+hashIndex[threadIdx.x]].x;



// for( candidate[threadIdx.x]=gstart[threadIdx.x]; candidate[threadIdx.x]<gstart[threadIdx.x]+ghit_num[threadIdx.x];candidate[threadIdx.x]++){


//cout<<"candidate[threadIdx.x]   "<<candidate[threadIdx.x]<<" hashIndex[threadIdx.x] "<<hashIndex[threadIdx.x]<<" hit_hash[hashIndex[threadIdx.x]].end "<<hit_hash[hashIndex[threadIdx.x]].end<<endl;







	



						 
		// track_index[threadIdx.x]=manager[cnum[threadIdx.x]+1];

	track[threadIdx.x]=&localtrack[threadIdx.x];


	

	small_reset(track[threadIdx.x]);





					selected_hit[threadIdx.x].x=position[(sblockIdx[threadIdx.x])*max_hit+candidate[threadIdx.x]].x;
					selected_hit[threadIdx.x].y=position[(sblockIdx[threadIdx.x])*max_hit+candidate[threadIdx.x]].y;
					selected_hit[threadIdx.x].z=position[(sblockIdx[threadIdx.x])*max_hit+candidate[threadIdx.x]].z;


					
					

					  r2[threadIdx.x]           =square(selected_hit[threadIdx.x].x)+square(selected_hit[threadIdx.x].y) ;


					 
				   // r[threadIdx.x]             = (float)sqrt ( r2[threadIdx.x] ) ;
				  selected_hit[threadIdx.x].phi           = (float)atan2(selected_hit[threadIdx.x].y,selected_hit[threadIdx.x].x) + getPara_phiShift ;
				  if (selected_hit[threadIdx.x].phi < 0 ) selected_hit[threadIdx.x].phi = selected_hit[threadIdx.x].phi + twoPi ;
				  // l3Log("r: %f, z: %f\n",r, selected_hit[threadIdx.x].z);
				  selected_hit[threadIdx.x].eta           = (float)seta((float)sqrt ( r2[threadIdx.x] ),selected_hit[threadIdx.x].z) ;

				  if ( getPara_szFitFlag ) {
					selected_hit[threadIdx.x].s  = 0.F ;
					selected_hit[threadIdx.x].wz = (float)(1./ square ( getPara_szErrorScale *0.24 ));
				  }


				  
				  
				  
				  

					 invR2[threadIdx.x]        = 1.F / r2[threadIdx.x] ;
					selected_hit[threadIdx.x].xp    =     selected_hit[threadIdx.x].x * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].yp    =   - selected_hit[threadIdx.x].y * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].wxy   =   r2[threadIdx.x] * r2[threadIdx.x] /  ( square(getPara_xyErrorScale)
					* ( square(0.12) + square(0.12) ) ) ;
					selected_hit[threadIdx.x].id= candidate[threadIdx.x];
					selected_hit[threadIdx.x].row=rhoi[threadIdx.x];








	small_add(&selected_hit[threadIdx.x],GO_DOWN,track[threadIdx.x]);


//position[(sblockIdx[threadIdx.x])*max_hit+candidate[threadIdx.x]].w=-1;


candidate[threadIdx.x]=0;





//		cout<<" smhit[candidate[threadIdx.x]] "<<smhit[candidate[threadIdx.x]].id<<endl;


	 cei[threadIdx.x]=etai[threadIdx.x];
	 cpi[threadIdx.x]=phii[threadIdx.x];
	 cri[threadIdx.x]=rhoi[threadIdx.x];
//cout<<" bug1 "<<hashIndex[threadIdx.x]<<"  "<<etai[threadIdx.x]<<"  "<<phii[threadIdx.x]<<"  "<<rhoi[threadIdx.x]<<"  "<<track[threadIdx.x]->nHits<<"  "<<candidate[threadIdx.x]<<endl;


 miss[threadIdx.x]=0;
	for( ccri[threadIdx.x]=cri[threadIdx.x]-1;ccri[threadIdx.x]>0;ccri[threadIdx.x]--){

	








	if(track[threadIdx.x]->nHits >= getPara_nHitsForSegment)break;
		track[threadIdx.x]->chi2[0] = getPara_maxDistanceSegment ;
		
		

		 result[threadIdx.x]=0;



		for( map_index[threadIdx.x]=0;map_index[threadIdx.x]<9;map_index[threadIdx.x]++){
			
				
				


				 ccei[threadIdx.x]=cei[threadIdx.x]+loop_eta[map_index[threadIdx.x]];
				 ccpi[threadIdx.x]=cpi[threadIdx.x]+loop_phi[map_index[threadIdx.x]];

				if(ccei[threadIdx.x]>40||ccei[threadIdx.x]<1) continue;
				if(ccpi[threadIdx.x]>10||ccpi[threadIdx.x]<1) continue;

 				 cchashIndex[threadIdx.x] =  (ccri[threadIdx.x])  * (41*11) + 
				(ccpi[threadIdx.x]) * 41 + (ccei[threadIdx.x]) ;
//		 cout<<" ccri[threadIdx.x] "<<ccri[threadIdx.x]<<" ccpi[threadIdx.x] "<<ccpi[threadIdx.x]<<"ccei[threadIdx.x] "<<ccei[threadIdx.x]<<endl;

				//maplist[threadIdx.x][map_index[threadIdx.x]]=map[cchashIndex[threadIdx.x]];

				
				
				 hit_num[threadIdx.x]=hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].y-hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].x;
				 start[threadIdx.x]=hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].x;
				
				

				for( hl[threadIdx.x]=0;hl[threadIdx.x]<hit_num[threadIdx.x];hl[threadIdx.x]++){





					selected_hit[threadIdx.x].x=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].x;
					selected_hit[threadIdx.x].y=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].y;
					selected_hit[threadIdx.x].z=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].z;



					  r2[threadIdx.x]           =square(selected_hit[threadIdx.x].x)+square(selected_hit[threadIdx.x].y) ;


					 
				   // r[threadIdx.x]             = (float)sqrt ( r2[threadIdx.x] ) ;
				  selected_hit[threadIdx.x].phi           = (float)atan2(selected_hit[threadIdx.x].y,selected_hit[threadIdx.x].x) + getPara_phiShift ;
				  if (selected_hit[threadIdx.x].phi < 0 ) selected_hit[threadIdx.x].phi = selected_hit[threadIdx.x].phi + twoPi ;
				  // l3Log("r: %f, z: %f\n",r, selected_hit[threadIdx.x].z);
				  selected_hit[threadIdx.x].eta           = (float)seta((float)sqrt ( r2[threadIdx.x] ),selected_hit[threadIdx.x].z) ;

				  if ( getPara_szFitFlag ) {
					selected_hit[threadIdx.x].s  = 0.F ;
					selected_hit[threadIdx.x].wz = (float)(1./ square ( getPara_szErrorScale *0.24 ));
				  }

				  
				  
				  
				  

					 invR2[threadIdx.x]        = 1.F / r2[threadIdx.x] ;
					selected_hit[threadIdx.x].xp    =     selected_hit[threadIdx.x].x * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].yp    =   - selected_hit[threadIdx.x].y * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].wxy   =   r2[threadIdx.x] * r2[threadIdx.x] /  ( square(getPara_xyErrorScale)
					* ( square(0.12) + square(0.12) ) ) ;
					selected_hit[threadIdx.x].id=start[threadIdx.x]+hl[threadIdx.x];
					selected_hit[threadIdx.x].row=ccri[threadIdx.x];


				
					 
				
							result[threadIdx.x]= small_segmentHitSelection(&track[threadIdx.x]->lasthit,&selected_hit[threadIdx.x],track[threadIdx.x]);

				
							
							
							
								 if ( result[threadIdx.x] > 0 ) {
										pos_index[threadIdx.x]=start[threadIdx.x]+hl[threadIdx.x];

							if ( result[threadIdx.x] ==2  ) break ; 
						 }				
				}



	
	


             if ( result[threadIdx.x] > 0 ) {
				curmap[threadIdx.x]=map_index[threadIdx.x];
				curpos[threadIdx.x]=pos_index[threadIdx.x];
                if ( result[threadIdx.x] ==2  ) break; ; 
             }


		}
//cout<<" result[threadIdx.x] "<<result[threadIdx.x]<<endl;
			

		if(result[threadIdx.x]>0){
			miss[threadIdx.x]=0;
			//element_setbit(maplist[threadIdx.x][curmap[threadIdx.x]],curpos[threadIdx.x]);
			if ( getPara_szFitFlag  ){
				 dx[threadIdx.x] = selected_hit[threadIdx.x].x - track[threadIdx.x]->lasthit.x ;
				 dy[threadIdx.x] = selected_hit[threadIdx.x].y - track[threadIdx.x]->lasthit.y ;
            track[threadIdx.x]->length    += (float)sqrt ( dx[threadIdx.x] * dx[threadIdx.x] + dy[threadIdx.x] * dy[threadIdx.x] ) ;
            selected_hit[threadIdx.x].s      = track[threadIdx.x]->length ;
         }
			small_add(&selected_hit[threadIdx.x],GO_DOWN,track[threadIdx.x]);

			// if(position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w<0){candidate[threadIdx.x]++;}else{
			// candidate[threadIdx.x]=0;
			// }
			// if(candidate[threadIdx.x]>2) break;

			
			cei[threadIdx.x]=cei[threadIdx.x]+loop_eta[curmap[threadIdx.x]];
			cpi[threadIdx.x]=cpi[threadIdx.x]+loop_phi[curmap[threadIdx.x]];
			cri[threadIdx.x]=ccri[threadIdx.x];
 				 cchashIndex[threadIdx.x] =  (ccri[threadIdx.x])  * (41*11) + 
				(cpi[threadIdx.x]) * 41 + (cei[threadIdx.x]) ;
				//map[cchashIndex[threadIdx.x]]=maplist[threadIdx.x][curmap[threadIdx.x]];
				position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w=-1;

		}else{
			miss[threadIdx.x]++;
			if(miss[threadIdx.x]>=2)	break;		
		}
		
		
	
	
	}




	if(track[threadIdx.x]->nHits < getPara_nHitsForSegment||candidate[threadIdx.x]>2){ 
	// cnum[threadIdx.x]=atomicSub(&manager[0],1);
	// manager[cnum[threadIdx.x]]=track_index[threadIdx.x];
	continue;}

//cout<<" track[threadIdx.x]->nHits "<<track[threadIdx.x]->nHits<<endl;


    xyChi2[threadIdx.x] = track[threadIdx.x]->chi2[0] ;
    szChi2[threadIdx.x] = track[threadIdx.x]->chi2[1] ;


	for( ccri[threadIdx.x]=cri[threadIdx.x]-1;ccri[threadIdx.x]>getPara_rowInnerMost;ccri[threadIdx.x]--){

		
	
		track[threadIdx.x]->chi2[0] = getPara_hitChi2Cut ;
		
		 result[threadIdx.x]=0;

//cout<<" bug2 "<<track_index[threadIdx.x]<<endl;

		for( map_index[threadIdx.x]=0;map_index[threadIdx.x]<9;map_index[threadIdx.x]++){
			
				
				


				 ccei[threadIdx.x]=cei[threadIdx.x]+loop_eta[map_index[threadIdx.x]];
				 ccpi[threadIdx.x]=cpi[threadIdx.x]+loop_phi[map_index[threadIdx.x]];

				if(ccei[threadIdx.x]>40||ccei[threadIdx.x]<1) continue;
				if(ccpi[threadIdx.x]>10||ccpi[threadIdx.x]<1) continue;
//		 cout<<"follow  ccri[threadIdx.x] "<<ccri[threadIdx.x]<<" ccpi[threadIdx.x] "<<ccpi[threadIdx.x]<<" ccei[threadIdx.x] "<<ccei[threadIdx.x]<<endl;



 				 cchashIndex[threadIdx.x] =  (ccri[threadIdx.x])  * (41*11) + 
				(ccpi[threadIdx.x]) * 41 + (ccei[threadIdx.x]) ;

				//maplist[threadIdx.x][map_index[threadIdx.x]]=map[cchashIndex[threadIdx.x]];

				 hit_num[threadIdx.x]=hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].y-hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].x;
				 start[threadIdx.x]=hash[(sblockIdx[threadIdx.x])*HASHSIZE+cchashIndex[threadIdx.x]].x;



				

				for( hl[threadIdx.x]=0;hl[threadIdx.x]<hit_num[threadIdx.x];hl[threadIdx.x]++){








					selected_hit[threadIdx.x].x=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].x;
					selected_hit[threadIdx.x].y=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].y;
					selected_hit[threadIdx.x].z=position[(sblockIdx[threadIdx.x])*max_hit+start[threadIdx.x]+hl[threadIdx.x]].z;



					 // x[threadIdx.x]            = selected_hit[threadIdx.x].x - getPara_xVertex ;
					 // y[threadIdx.x]            = selected_hit[threadIdx.x].y - getPara_yVertex ;
					 // r2[threadIdx.x]           = x[threadIdx.x] * x[threadIdx.x] + y[threadIdx.x] * y[threadIdx.x] ;
					  r2[threadIdx.x]           =square(selected_hit[threadIdx.x].x)+square(selected_hit[threadIdx.x].y) ;


					 
				   // r[threadIdx.x]             = (float)sqrt ( r2[threadIdx.x] ) ;
				  selected_hit[threadIdx.x].phi           = (float)atan2(selected_hit[threadIdx.x].y,selected_hit[threadIdx.x].x) + getPara_phiShift ;
				  if (selected_hit[threadIdx.x].phi < 0 ) selected_hit[threadIdx.x].phi = selected_hit[threadIdx.x].phi + twoPi ;
				  // l3Log("r: %f, z: %f\n",r, selected_hit[threadIdx.x].z);
				  selected_hit[threadIdx.x].eta           = (float)seta((float)sqrt ( r2[threadIdx.x] ),selected_hit[threadIdx.x].z) ;

				  if ( getPara_szFitFlag ) {
					selected_hit[threadIdx.x].s  = 0.F ;
					selected_hit[threadIdx.x].wz = (float)(1./ square ( getPara_szErrorScale *0.24 ));
				  }


				  
				  
				  
				  

					 invR2[threadIdx.x]        = 1.F / r2[threadIdx.x] ;
					selected_hit[threadIdx.x].xp    =     selected_hit[threadIdx.x].x * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].yp    =   - selected_hit[threadIdx.x].y * invR2[threadIdx.x] ;
					selected_hit[threadIdx.x].wxy   =   r2[threadIdx.x] * r2[threadIdx.x] /  ( square(getPara_xyErrorScale)
					* ( square(0.12) + square(0.12) ) ) ;
					selected_hit[threadIdx.x].id=start[threadIdx.x]+hl[threadIdx.x];
					selected_hit[threadIdx.x].row=ccri[threadIdx.x];





result[threadIdx.x]= small_followHitSelection(&track[threadIdx.x]->lasthit,&selected_hit[threadIdx.x],GO_DOWN,track[threadIdx.x]);
		         





				 if ( result[threadIdx.x] > 0 ) {
							pos_index[threadIdx.x]=start[threadIdx.x]+hl[threadIdx.x];;


                if ( result[threadIdx.x] ==2  ) break ; 
             }
	
					
				
				}







				
				
				
				
				
				
				
             if ( result[threadIdx.x] > 0 ) {
				curmap[threadIdx.x]=map_index[threadIdx.x];
				curpos[threadIdx.x]=pos_index[threadIdx.x];
                if ( result[threadIdx.x] ==2  ) break; ; 
             }


		}
//cout<<" bug3 "<<track_index[threadIdx.x]<<endl;
		if(result[threadIdx.x]>0){
			miss[threadIdx.x]=0;
			//element_setbit(maplist[threadIdx.x][curmap[threadIdx.x]],curpos[threadIdx.x]);
			



       lxyChi2[threadIdx.x] = track[threadIdx.x]->chi2[0]-track[threadIdx.x]->chi2[1] ;
      xyChi2[threadIdx.x] += lxyChi2[threadIdx.x] ;
      selected_hit[threadIdx.x].xyChi2 = lxyChi2[threadIdx.x] ;
//
//   if sz fit update track[threadIdx.x] length
//
      if ( getPara_szFitFlag  ) {
         track[threadIdx.x]->length = selected_hit[threadIdx.x].s ;
         szChi2[threadIdx.x] += track[threadIdx.x]->chi2[1]  ;
         selected_hit[threadIdx.x].szChi2 = track[threadIdx.x]->chi2[1] ;

      }
//
//     Add hit to track[threadIdx.x]
//
			small_add(&selected_hit[threadIdx.x],GO_DOWN,track[threadIdx.x]);

			
			// if(position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w<0){candidate[threadIdx.x]++;}else{
			// candidate[threadIdx.x]=0;
			// }
			// if(candidate[threadIdx.x]>2) break;
			
			
//cout<<" bug4 "<<track[threadIdx.x]->nHits<<endl;
	
			
			// if(track[threadIdx.x]->nHits == getPara_minHitsPerTrack){
			
		// for( map_index[threadIdx.x]=0;map_index[threadIdx.x]<getPara_minHitsPerTrack;map_index[threadIdx.x]++){
// localtrack[threadIdx.x].hitmap[map_index[threadIdx.x]];	
// position[(sblockIdx[threadIdx.x])*max_hit+localtrack[threadIdx.x].hitmap[map_index[threadIdx.x]]].w=-1;		
	// }
	// }else if(track[threadIdx.x]->nHits > getPara_minHitsPerTrack){
	
// position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w=-1;	
	// }			
			
			// position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w=-1;	
			


		
			
			
			
			cei[threadIdx.x]=cei[threadIdx.x]+loop_eta[curmap[threadIdx.x]];
			cpi[threadIdx.x]=cpi[threadIdx.x]+loop_phi[curmap[threadIdx.x]];
			cri[threadIdx.x]=ccri[threadIdx.x];
 				 cchashIndex[threadIdx.x] =  (ccri[threadIdx.x])  * (41*11) + 
				(cpi[threadIdx.x]) * 41 + (cei[threadIdx.x]) ;
				// map[cchashIndex[threadIdx.x]]=maplist[threadIdx.x][curmap[threadIdx.x]];
				position[(sblockIdx[threadIdx.x])*max_hit+curpos[threadIdx.x]].w=-1;

		}else{
			miss[threadIdx.x]++;
			if(miss[threadIdx.x]>=3)	break;		
		}
		
		
	
	
	}


   if ( track[threadIdx.x]->nHits < getPara_minHitsPerTrack||candidate[threadIdx.x]>2 ) { 
   // cnum[threadIdx.x]=atomicSub(&manager[0],1);
   // manager[cnum[threadIdx.x]]=track_index[threadIdx.x];
   continue ;} 
//
//   Store track[threadIdx.x] chi2
//

	
   track[threadIdx.x]->chi2[0] = xyChi2[threadIdx.x] ;
   track[threadIdx.x]->chi2[1] = szChi2[threadIdx.x] ;
//
//        Check total chi2
//
   float normalized_chi2 = (track[threadIdx.x]->chi2[0]+track[threadIdx.x]->chi2[1])/track[threadIdx.x]->nHits ;
   if ( normalized_chi2 > getPara_trackChi2Cut ) {
   // cnum[threadIdx.x]=atomicSub(&manager[0],1);
   // manager[cnum[threadIdx.x]]=track_index[threadIdx.x];
   continue ;}	

   
cnum[threadIdx.x]=atomicAdd(&manager[sblockIdx[threadIdx.x]],1);   
   
   
// out[track_index[threadIdx.x]]=localtrack[threadIdx.x].nHits;	


 out_track[cnum[threadIdx.x]].nHits=localtrack[threadIdx.x].nHits;
 out_track[cnum[threadIdx.x]].lastXyAngle=localtrack[threadIdx.x].lastXyAngle;	
 out_track[cnum[threadIdx.x]].s11Xy=localtrack[threadIdx.x].s11Xy;	
 out_track[cnum[threadIdx.x]].s12Xy=localtrack[threadIdx.x].s12Xy;	
 out_track[cnum[threadIdx.x]].s22Xy=localtrack[threadIdx.x].s22Xy;	
 out_track[cnum[threadIdx.x]].g1Xy=localtrack[threadIdx.x].g1Xy;	
 out_track[cnum[threadIdx.x]].g2Xy=localtrack[threadIdx.x].g2Xy;	
 out_track[cnum[threadIdx.x]].s11Sz=localtrack[threadIdx.x].s11Sz;	
 out_track[cnum[threadIdx.x]].s12Sz=localtrack[threadIdx.x].s12Sz;	
 out_track[cnum[threadIdx.x]].s22Sz=localtrack[threadIdx.x].s22Sz;	
 out_track[cnum[threadIdx.x]].g1Sz=localtrack[threadIdx.x].g1Sz;	
 out_track[cnum[threadIdx.x]].g2Sz=localtrack[threadIdx.x].g2Sz;	
 out_track[cnum[threadIdx.x]].ddXy=localtrack[threadIdx.x].ddXy;	
 out_track[cnum[threadIdx.x]].a1Xy=localtrack[threadIdx.x].a1Xy;	
 out_track[cnum[threadIdx.x]].a2Xy=localtrack[threadIdx.x].a2Xy;	
 out_track[cnum[threadIdx.x]].a2Sz=localtrack[threadIdx.x].a2Sz;	
 out_track[cnum[threadIdx.x]].a1Sz=localtrack[threadIdx.x].a1Sz;	
 out_track[cnum[threadIdx.x]].chi2[0]=localtrack[threadIdx.x].chi2[0];	
 out_track[cnum[threadIdx.x]].chi2[1]=localtrack[threadIdx.x].chi2[1];	
 out_track[cnum[threadIdx.x]].length=localtrack[threadIdx.x].length;	
 

		


		for( map_index[threadIdx.x]=0;map_index[threadIdx.x]<localtrack[threadIdx.x].nHits;map_index[threadIdx.x]++){
	out_track[cnum[threadIdx.x]].hitmap[map_index[threadIdx.x]]=localtrack[threadIdx.x].hitmap[map_index[threadIdx.x]];	
	}
	

 }while(1);




 


				// }
				// }}
				
 //__syncthreads();				
 //out[sindex[threadIdx.x]]=manager[sblockIdx[threadIdx.x]];				
			
			









	




}






__device__ float dtest(float *data){


return 0;
}





__device__ int small_segmentHitSelection ( small_FtfHit *baseHit, small_FtfHit *candidateHit,small_FtfTrack *tra ){


   float dx, dy, dr, d3, dangle ;
   float dphi, deta ;
   float   angle ;
  
//
//   select hit with the
//   the smallest value of d3 (defined below)
//
   dphi  = (float)fabs((baseHit->phi) - (candidateHit->phi)) ; 
   if ( dphi > pi ) dphi = (float)fabs( twoPi - dphi ) ;
   if ( dphi > getPara_dphi && dphi < twoPi -getPara_dphi ) return 0 ;
//
//    Make sure we want to look at the difference in eta
//
   if ( baseHit->dz < 1000. && candidateHit->dz < 1000. ){
        deta  = (float)fabs((baseHit->eta) - (candidateHit->eta)) ; 
        if ( deta > getPara_deta ) return 0 ;
   }
   else deta = 0.F ;
  
   dr    = (float)fabs((float)(baseHit->row - candidateHit->row));
   d3    = (float)(toDeg * dr * ( dphi  + deta ) ) ;
//
//     If initial segment is longer than 2 store angle info in 
//     a1Xy and a1_sz
//

  
   if ( getPara_nHitsForSegment > 2 && tra->nHits-1 < getPara_nHitsForSegment ) {
	  ;
      dx = candidateHit->x - baseHit->x ;
      dy = candidateHit->y - baseHit->y ;
      angle = (float)atan2 ( dy, dx ) ;
	 
      if ( angle < 0  ) angle = angle + twoPi ;
      tra->lastXyAngle = angle ;
   }

   if ( d3 < tra->chi2[0] ) {
//
//   For second hit onwards check the difference in angle 
//   between the last two track segments
//
    
      if ( tra->nHits > 1 ) {
	 dx     = candidateHit->x - baseHit->x ;
         dy     = candidateHit->y - baseHit->y ;
         angle  = (float)atan2 ( dy, dx ) ;
         if ( angle < 0  ) angle = angle + twoPi ;
	    dangle = (float)fabs ( tra->lastXyAngle - angle );
		  
	    tra->lastXyAngle = angle ;
         if ( dangle > getPara_segmentMaxAngle ) return 0 ;
      }
//
//    Check whether this is the "closest" hit
//
      tra->chi2[0]          = d3 ;
      if ( d3 < getPara_goodDistance ) return 2 ;
	  return 1 ;
   }
//
//    If hit does not fulfill criterai return 0
//
   return 0 ;
}


__device__ int small_followHitSelection ( small_FtfHit *baseHit, small_FtfHit *candidateHit,  int way, small_FtfTrack *tra ){
//
   float lszChi2 = 0 ;
   float lchi2 ;
   float slocal=0, deta, dphi ;
   float dx, dy, dxy, dsz, temp ;
//
//           Check delta eta 
//
//   if ( baseHit->dz < 1000. && candidateHit->dz < 1000 ){
      deta = fabs((baseHit->eta)-(candidateHit->eta)) ;
      if ( deta > getPara_deta ) return 0 ; 
//   }
//   else deta = 0.F ;
//
//           Check delta phi
//
  dphi = fabs((baseHit->phi)-(candidateHit->phi)) ;
  if ( dphi > getPara_dphi && dphi < twoPi-getPara_dphi ) return 0 ;
//
//      If looking for secondaries calculate conformal coordinates
//

//
//      Calculate distance in x and y
//
   temp = (tra->a2Xy * candidateHit->xp - candidateHit->yp + tra->a1Xy) ;
   dxy  = temp * temp / ( tra->a2Xy * tra->a2Xy + 1.F ) ;
//
//    Calculate chi2
//
   lchi2    = (dxy * candidateHit->wxy) ;

   if ( lchi2 > tra->chi2[0] ) return 0 ;
//
//      Now in the sz plane
//
   if ( getPara_szFitFlag ){
//
//        Get "s" and calculate distance hit-line
//
      dx     = baseHit->x - candidateHit->x ;
      dy     = baseHit->y - candidateHit->y ;
      slocal = baseHit->s - way * sqrt ( dx * dx + dy * dy ) ;

      temp = (tra->a2Sz * slocal - candidateHit->z + tra->a1Sz) ;
      dsz  = temp * temp / ( tra->a2Sz * tra->a2Sz + 1 ) ;
//
//              Calculate chi2
//
      lszChi2 = dsz * candidateHit->wz ;
      lchi2 += lszChi2 ;
   } 
   else {
      lszChi2 = 0.F ;
      //slocal = 0;
   }
//
//         Check whether the chi2 square is better than previous one
//
   if ( lchi2 < tra->chi2[0] ) {
      tra->chi2[0]       = (float)lchi2    ;
      tra->chi2[1]       = (float)lszChi2 ;
      
      if ( getPara_szFitFlag  ) candidateHit->s = (float)slocal ;
//
//       if a good chi2 is found let's stop here
//
      if ( lchi2 < getPara_goodHitChi2 ) return 2 ;

      return 1 ;
   }
//
//     Return the selected hit
//
   return 0 ;
}






__device__ void small_add ( small_FtfHit *thisHit, int way, small_FtfTrack *tra  )
{

//	cout<<"small_add  "<<thisHit->id<<endl;
//
//      Increment # hits in this track
//
  if(tra->nHits>=50) return;
  tra->hitmap[tra->nHits]=thisHit->id;


	tra->nHits++ ; 
	tra->lasthit.phi=	thisHit->phi;
	tra->lasthit.dz=	thisHit->dz;
	tra->lasthit.eta=	thisHit->eta;
	tra->lasthit.row=	thisHit->row;
	tra->lasthit.x=	thisHit->x;
	tra->lasthit.y=	thisHit->y;
	tra->lasthit.s=	thisHit->s;

	

//
//    Check whether a fit update is needed
//
  if ( tra->nHits < getPara_minHitsForFit ) return ;
//
//    Include hit in xy fit parameter calculation
//
  
  //if(thisHit->id==3584){
  //cout<<tra->s11Xy<<endl;
  //cout<<tra->s12Xy<<endl;
  //cout<<tra->s22Xy<<endl;
  //cout<<tra->g1Xy<<endl;
  //cout<<tra->g2Xy<<endl;
  //cout<<tra->a1Xy<<endl;
  //cout<<tra->a2Xy<<endl;
  //cout<<tra->a1Sz<<endl;
  //cout<<tra->a2Sz<<endl;
  //cout<<thisHit->wxy<<endl;
  //cout<<thisHit->xp<<endl;
  //cout<<thisHit->yp<<endl;
  //cout<<thisHit->s<<endl;
  //cout<<thisHit->z<<endl;
  //cout<<tra->nHits<<endl;

  //}


  tra->s11Xy = tra->s11Xy + thisHit->wxy ;
  tra->s12Xy = tra->s12Xy + thisHit->wxy * thisHit->xp ;
  tra->s22Xy = tra->s22Xy + thisHit->wxy * square(thisHit->xp) ;
  tra->g1Xy  = tra->g1Xy  + thisHit->wxy * thisHit->yp ;
  tra->g2Xy  = tra->g2Xy  + thisHit->wxy * thisHit->xp * thisHit->yp ;
  
 
  if ( tra->nHits > getPara_minHitsForFit  )
  {
     tra->ddXy  = tra->s11Xy * tra->s22Xy - square ( tra->s12Xy ) ;
     if ( tra->ddXy != 0 ) {
        tra->a1Xy  = ( tra->g1Xy * tra->s22Xy - tra->g2Xy * tra->s12Xy ) / tra->ddXy ;
        tra->a2Xy  = ( tra->g2Xy * tra->s11Xy - tra->g1Xy * tra->s12Xy ) / tra->ddXy ;
     }
     else {
		 //LOG(ERR, "FtfTrack:add: ddSz = 0 \n" ) ;
     }
  }
//
//     Now in the sz plane
//
  if ( getPara_szFitFlag ) {
     tra->s11Sz = tra->s11Sz + thisHit->wz ;
     tra->s12Sz = tra->s12Sz + thisHit->wz * thisHit->s ;
     tra->s22Sz = tra->s22Sz + thisHit->wz * thisHit->s * thisHit->s ;
     tra->g1Sz  = tra->g1Sz  + thisHit->wz * thisHit->z ;
     tra->g2Sz  = tra->g2Sz  + thisHit->wz * thisHit->s * thisHit->z ;
  
     if ( tra->nHits > getPara_minHitsForFit ) {
		
        tra->ddSz  = tra->s11Sz * tra->s22Sz -  tra->s12Sz * tra->s12Sz ;
	if ( tra->ddSz != 0 ) {
           tra->a1Sz  = ( tra->g1Sz * tra->s22Sz - tra->g2Sz * tra->s12Sz ) / tra->ddSz ;
           tra->a2Sz  = ( tra->g2Sz * tra->s11Sz - tra->g1Sz * tra->s12Sz ) / tra->ddSz ;
         }
         else
         {
            if ( getPara_infoLevel > 0 ) {
               //LOG(ERR, "FtfTrack:add: ddSz = 0 \n" ) ;
            }
         }
      }
   }
}




__device__ void small_reset ( small_FtfTrack *tra)
{
/*----------------------------------------------------------------------
                Set fit parameters to zero
----------------------------------------------------------------------*/
 
  //tra->flag     = getPara_primaries ;


  tra->nHits    = 0 ;
  tra->s11Xy   = 
  tra->s12Xy   = 
  tra->s22Xy   = 
  tra->g1Xy    = 
  tra->g2Xy    = 
  tra->chi2[0]  = 0.F ;
 
  //tra->nxatrk   = 0 ;
  if ( getPara_szFitFlag ) 
  {
     tra->s11Sz =
     tra->s12Sz =
     tra->s22Sz =
     tra->g1Sz  =
     tra->g2Sz  =
     tra->chi2[1]  = 
     tra->length         = 0.F ;
  }


}


// __device__ void small_track_manager::ini(){
	// for(int i=0;i<2000;i++){
		// this->track_id[i]=i;
	// }
	// this->maxused=0;
	// this->number=0;

// }




//void small_track_assign::ini(){
//
//	this->number=0;
//
//}
//
//
//int small_track_assign::push(int id){
//
//track_id[number]=id;
//number++;
//return number;
//
//}




__device__ int element_setbit(int & data,int position){
if(position>31) return -1;

unsigned short mask=0;
mask=1<<position;
data=data|mask;
return 1;

}


__device__ int element_getbit(int data,int position){

int sd=data>>position;
return sd%2;

}



__device__ int small_segmentHitGroup ( small_FtfHit *candidateHit,int num,small_FtfHit *selected_hit,int &selected_pos, small_FtfTrack *tra ){
		int result=0;
	for(int i=0;i<num;i++){
				result= small_segmentHitSelection(&tra->lasthit,&candidateHit[i],tra);
		             if ( result > 0 ) {
							selected_pos=i;
						   selected_hit->id=candidateHit[i].id;
						   selected_hit->row=candidateHit[i].row;

						   selected_hit->x=candidateHit[i].x;
						   selected_hit->y=candidateHit[i].y;
						   selected_hit->z=candidateHit[i].z;
						   selected_hit->xp=candidateHit[i].xp;
						   selected_hit->yp=candidateHit[i].yp;
						   selected_hit->eta=candidateHit[i].eta;
						   selected_hit->phi=candidateHit[i].phi;
						   selected_hit->wxy=candidateHit[i].wxy;
						   selected_hit->wz=candidateHit[i].wz;

						   selected_hit->dx=candidateHit[i].dx;
						   selected_hit->dy=candidateHit[i].dy;
						   selected_hit->dz=candidateHit[i].dz;
						   selected_hit->s=candidateHit[i].s;


                if ( result ==2  ) return 2 ; 
             }
	
	}

return result;

}


__device__ int small_followHitGroup (  small_FtfHit *candidateHit,  int way, int num,small_FtfHit *selected_hit,int &selected_pos, small_FtfTrack *tra ){


		int result=0;
	for(int i=0;i<num;i++){
				result= small_followHitSelection(&tra->lasthit,&candidateHit[i],way,tra);
		             if ( result > 0 ) {
							selected_pos=i;
						   selected_hit->id=candidateHit[i].id;
						   selected_hit->row=candidateHit[i].row;

						   selected_hit->x=candidateHit[i].x;
						   selected_hit->y=candidateHit[i].y;
						   selected_hit->z=candidateHit[i].z;
						   selected_hit->xp=candidateHit[i].xp;
						   selected_hit->yp=candidateHit[i].yp;
						   selected_hit->eta=candidateHit[i].eta;
						   selected_hit->phi=candidateHit[i].phi;
						   selected_hit->wxy=candidateHit[i].wxy;
						   selected_hit->wz=candidateHit[i].wz;

						   selected_hit->dx=candidateHit[i].dx;
						   selected_hit->dy=candidateHit[i].dy;
						   selected_hit->dz=candidateHit[i].dz;
						   selected_hit->s=candidateHit[i].s;


                if ( result ==2  ) return 2 ; 
             }
	
	}

return result;


}







