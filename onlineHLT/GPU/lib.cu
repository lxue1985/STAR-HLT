#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include "lib_kernel.cuh"


#  define CUDA_SAFE_CALL_NO_SYNC( call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } } while (0)

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);                                            \


#include <cuda_gl_interop.h>

extern "C"
{



void allocateArray(void **devPtr, size_t size)
{
    CUDA_SAFE_CALL(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    CUDA_SAFE_CALL(hipFree(devPtr));
}

void threadSync()
{
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void copyArrayFromDevice(void* host, const void* device,  int size)
{   

    CUDA_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    CUDA_SAFE_CALL(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}


int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


void Gtest(float* position, int*hash,int*map,int* manager, float *out, int *number,int cores){

    // int numThreads, numBlocks;
    // computeGridSize(number, 16, numBlocks, numThreads);
	
	//cout<<numBlocks<<"  "<<numThreads<<endl;
	// printf("\n numBlocks %d  numThreads%d number %d \n",numBlocks,numThreads,number);
	// CUDA_SAFE_CALL(hipBindTexture(0, positionTex, position, 10000*sizeof(float4)));
    // CUDA_SAFE_CALL(hipBindTexture(0, hashTex, hash, HASHSIZE*sizeof(int2)));

test<<<cores,24>>>((float4*)position,(int2*)hash,map,manager,out,number);

    // CUDA_SAFE_CALL(hipUnbindTexture(positionTex));
    // CUDA_SAFE_CALL(hipUnbindTexture(hashTex));
}

// void Gcalc_hash(int number,float *fpara,float *fhit,int * volumeC,int * rowC){
    // int numThreads, numBlocks;
    // computeGridSize(number, 256, numBlocks, numThreads);
	// calc_hash<<< numBlocks, numThreads >>>(number,fpara,fhit,volumeC,rowC);
	

// }


// void Gclear(int * dataC,int number){
    // int numThreads, numBlocks;
    // computeGridSize(number, 256, numBlocks, numThreads);
	// clear<<< numBlocks, numThreads >>>(dataC,number);

// }

// void GSclass_hash(int number,float *fpara,float *fhit,int * volumeC,int * rowC){

	// class_hash<<<1,1>>>(number,fpara,fhit,volumeC,rowC);

// }

// void Gtracking(int number,float *fpara,float *fhit,int * ivolumeC,int * irowC,int * itrackC,float *ftrack){
    // int numThreads, numBlocks;
    // computeGridSize(number, 16, numBlocks, numThreads);
	
	// tracking<<<numBlocks, numThreads>>>( number,fpara,fhit, ivolumeC, irowC, itrackC, ftrack);
// }





}